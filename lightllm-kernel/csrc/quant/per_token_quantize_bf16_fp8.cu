#include "hip/hip_runtime.h"
#include "ops_common.h"
#include "reduce/sm70.cuh"


namespace lightllm {
namespace ops {

using namespace lightllm;

// CUDA kernel for per token quantization from BF16 to FP8
template<int32_t TPB>
__global__ void device_per_token_quant_bf16_to_fp8_general(
    const bf16_t* __restrict__ input,  // Input tensor in BF16 format
    fp8_e4m3_t* __restrict__ output,   // Output tensor in FP8 format
    fp32_t* __restrict__ scales,       // Output scales for each token
    const int64_t N
) {
    const int32_t bid = blockIdx.x;
    const int32_t tid = threadIdx.x;
    constexpr fp32_t FP8_E4M3_MAX = 448.0f; // Maximum value representable in FP8 E4M3 format
    
    const bf16_t* _input = input + bid * N; // Input pointer for the token
    fp8_e4m3_t* _output  = output + bid * N; // Output pointer for the token

    fp32_t* _scales;
    _scales = scales + bid;

    // Local arrays for intermediate storage
    fp8_e4m3_t local_f8;
    bf16_t local_bf16;

    extern __shared__ bf16_t workspace1[];

    fp32_t local_max = -FLT_MAX;
    for (int32_t i = tid; i < N; i += TPB) {
        local_bf16 = _input[i];
        workspace1[i] = local_bf16;

        fp32_t tmp = cvt_bf16_f32(local_bf16);
        local_max = fmaxf(local_max, fabsf(tmp));
    }

    // Reduce the maximum value across the block
    const fp32_t reduced_max = lightllm::reduce::sm70::sync_block_reduce_max_f32<TPB>(local_max);

    // Compute the scale factor with epsilon to avoid division by zero
    constexpr fp32_t epsilon = 1e-7f;
    const fp32_t scale = reduced_max / FP8_E4M3_MAX;
    const fp32_t inv_scale = 1.0f / (scale + epsilon);

    for (int32_t i = tid; i < N; i += TPB) {
        local_bf16 = workspace1[i];
        
        fp32_t tmp = cvt_bf16_f32(local_bf16);
        fp32_t x = tmp * inv_scale;
        local_f8 = fp8_e4m3_t(x);

        _output[i] = local_f8;
    }

    if(tid == 0){
        *_scales = scale;
    }

}

// CUDA kernel for per token quantization from BF16 to FP8
template<int32_t TPB>
__global__ void device_per_token_quant_bf16_to_fp8_vpt(
    const bf16_t* __restrict__ input,  // Input tensor in BF16 format
    fp8_e4m3_t* __restrict__ output,   // Output tensor in FP8 format
    fp32_t* __restrict__ scales,       // Output scales for each token
    const int32_t N
) {
    constexpr int32_t VPT = 8;

    const int32_t bid = blockIdx.x;
    const int32_t tid = threadIdx.x;
    constexpr fp32_t FP8_E4M3_MAX = 448.0f; // Maximum value representable in FP8 E4M3 format
    
    const bf16_t* _input = input + bid * N; // Input pointer for the token
    fp8_e4m3_t* _output  = output + bid * N; // Output pointer for the token

    fp32_t* _scales;
     _scales = scales + bid;

    // Local arrays for intermediate storage
    fp8x4_e4m3_t local_f8[VPT / 4];
    bf16x2_t local_bf16[VPT / 2];

    extern __shared__ bf16x2_t workspace2[];

    fp32_t local_max = -FLT_MAX;
    for (int32_t i = tid * VPT; i < N; i += TPB * VPT) {
        // Load VPT FP16 elements from global memory (_X) into local vector (local_x).
        vec_copy<sizeof(bf16_t) * VPT>(_input + i, local_bf16);

        vec_copy<sizeof(bf16_t) * VPT>(local_bf16, workspace2 + (i >> 1));

        // Compute the max for the VPT elements.
        #pragma unroll
        for(int32_t j = 0; j< VPT/2; j++){
            fp32x2_t tmp = bf16x2_to_fp32x2(local_bf16[j]);
            fp32_t max = fmaxf(fabsf(tmp.x), fabsf(tmp.y));
            local_max = fmaxf(local_max, max);
        }
    }

    // Reduce the maximum value across the block
    const fp32_t reduced_max = lightllm::reduce::sm70::sync_block_reduce_max_f32<TPB>(local_max);

    // Compute the scale factor with epsilon to avoid division by zero
    constexpr fp32_t epsilon = 1e-7f;
    const fp32_t scale = reduced_max / FP8_E4M3_MAX;
    const fp32_t inv_scale = 1.0f / (scale + epsilon);

    for (int32_t i = tid * VPT; i < N; i += TPB * VPT) {
        vec_copy<sizeof(bf16_t) * VPT>(workspace2 + (i >> 1), local_bf16);

        #pragma unroll
        for (int32_t j = 0; j < VPT/4; j++) {
            fp32x2_t x = bf16x2_to_fp32x2(local_bf16[2 * j + 0]);
            fp32x2_t y = bf16x2_to_fp32x2(local_bf16[2 * j + 1]);
            fp32x4_t ret = make_float4(
                x.x * inv_scale,
                x.y * inv_scale,
                y.x * inv_scale,
                y.y * inv_scale
            );
            local_f8[j] = fp8x4_e4m3_t(ret);
        }

        vec_copy<sizeof(fp8_e4m3_t) * VPT>(local_f8, _output + i);
    }

    if(tid == 0){
        *_scales = scale;
    }
}



// CUDA kernel for per token quantization from BF16 to FP8
template<int32_t TPB, int32_t N>
__global__ void device_per_token_quant_bf16_to_fp8(
    const bf16_t* __restrict__ input,  // Input tensor in BF16 format
    fp8_e4m3_t* __restrict__ output,   // Output tensor in FP8 format
    fp32_t* __restrict__ scales       // Output scales for each token
) {
    constexpr int32_t VPT = 8;

    static_assert(N % 2 == 0, "N must be even.");
    static_assert(N % VPT == 0, "N must be a multiple of VPT.");

    const int32_t bid = blockIdx.x;
    const int32_t tid = threadIdx.x;
    constexpr fp32_t FP8_E4M3_MAX = 448.0f; // Maximum value representable in FP8 E4M3 format
    
    const bf16_t* _input = input + bid * N; // Input pointer for the token
    fp8_e4m3_t* _output  = output + bid * N; // Output pointer for the token

    fp32_t* _scales;
    _scales = scales + bid;

    // Local arrays for intermediate storage
    fp8x4_e4m3_t local_f8[VPT / 4];
    bf16x2_t local_bf16[VPT / 2];

    __shared__ bf16x2_t workspace[N / 2];

    fp32_t local_max = -FLT_MAX;
    for (int32_t i = tid * VPT; i < N; i += TPB * VPT) {
        // Load VPT FP16 elements from global memory (_X) into local vector (local_x).
        vec_copy<sizeof(bf16_t) * VPT>(_input + i, local_bf16);

        vec_copy<sizeof(bf16_t) * VPT>(local_bf16, workspace + (i >> 1));

        // Compute the max for the VPT elements.
        #pragma unroll
        for(int32_t j = 0; j< VPT/2; j++){
            fp32x2_t tmp = bf16x2_to_fp32x2(local_bf16[j]);
            fp32_t max = fmaxf(fabsf(tmp.x), fabsf(tmp.y));
            local_max = fmaxf(local_max, max);
        }
    }

    // Reduce the maximum value across the block
    const fp32_t reduced_max = lightllm::reduce::sm70::sync_block_reduce_max_f32<TPB>(local_max);

    // Compute the scale factor with epsilon to avoid division by zero
    constexpr fp32_t epsilon = 1e-7f;
    const fp32_t scale = reduced_max / FP8_E4M3_MAX;
    const fp32_t inv_scale = 1.0f / (scale + epsilon);

    for (int32_t i = tid * VPT; i < N; i += TPB * VPT) {
        vec_copy<sizeof(bf16_t) * VPT>(workspace + (i >> 1), local_bf16);

        #pragma unroll
        for (int32_t j = 0; j < VPT/4; j++) {
            fp32x2_t x = bf16x2_to_fp32x2(local_bf16[2 * j + 0]);
            fp32x2_t y = bf16x2_to_fp32x2(local_bf16[2 * j + 1]);
            fp32x4_t ret = make_float4(
                x.x * inv_scale,
                x.y * inv_scale,
                y.x * inv_scale,
                y.y * inv_scale
            );
            local_f8[j] = fp8x4_e4m3_t(ret);
        }

        vec_copy<sizeof(fp8_e4m3_t) * VPT>(local_f8, _output + i);
    }

    if(tid == 0){
        *_scales = scale;
    }
}


void per_token_quant_bf16_fp8 (
    Tensor& output,
    const Tensor& input,
    Tensor& scales
) {
    TORCH_CHECK(input.is_cuda(), "Input must be a CUDA tensor");
    TORCH_CHECK(input.dim() == 2, "Input must be 2-dimensional");
    TORCH_CHECK(input.scalar_type() == c10::kBFloat16, "Input must be BF16 type");

    Tensor contiguous_input = input.is_contiguous() ? input : input.contiguous();
    Tensor contiguous_scales = scales.is_contiguous() ? scales : scales.contiguous();

    const int64_t M = input.size(0);
    const int64_t N = input.size(1);

    const int32_t blocks = M;

    switch (N) {
        case 16:
            device_per_token_quant_bf16_to_fp8<128, 16>
            <<<blocks, 128, 0, at::cuda::getCurrentCUDAStream()>>>(
                PTR<bf16_t>(contiguous_input),
                PTR<fp8_e4m3_t>(output),
                PTR<fp32_t>(contiguous_scales)
            );
            break;
        case 32:
            device_per_token_quant_bf16_to_fp8<128, 32>
            <<<blocks, 128, 0, at::cuda::getCurrentCUDAStream()>>>(
                PTR<bf16_t>(contiguous_input),
                PTR<fp8_e4m3_t>(output),
                PTR<fp32_t>(contiguous_scales)
            );
            break;
        case 64:
            device_per_token_quant_bf16_to_fp8<128, 64>
            <<<blocks, 128, 0, at::cuda::getCurrentCUDAStream()>>>(
                PTR<bf16_t>(contiguous_input),
                PTR<fp8_e4m3_t>(output),
                PTR<fp32_t>(contiguous_scales)
            );
            break;
        case 512:
            device_per_token_quant_bf16_to_fp8<128, 512>
            <<<blocks, 128, 0, at::cuda::getCurrentCUDAStream()>>>(
                PTR<bf16_t>(contiguous_input),
                PTR<fp8_e4m3_t>(output),
                PTR<fp32_t>(contiguous_scales)
            );
            break;
        case 1024:
            device_per_token_quant_bf16_to_fp8<128, 1024>
            <<<blocks, 128, 0, at::cuda::getCurrentCUDAStream()>>>(
                PTR<bf16_t>(contiguous_input),
                PTR<fp8_e4m3_t>(output),
                PTR<fp32_t>(contiguous_scales)
            );
            break;
        case 3200:
            device_per_token_quant_bf16_to_fp8<128, 3200>
            <<<blocks, 128, 0, at::cuda::getCurrentCUDAStream()>>>(
                PTR<bf16_t>(contiguous_input),
                PTR<fp8_e4m3_t>(output),
                PTR<fp32_t>(contiguous_scales)
            );
            break;
        case 4096:
            device_per_token_quant_bf16_to_fp8<128, 4096>
            <<<blocks, 128, 0, at::cuda::getCurrentCUDAStream()>>>(
                PTR<bf16_t>(contiguous_input),
                PTR<fp8_e4m3_t>(output),
                PTR<fp32_t>(contiguous_scales)
            );
            break;
        case 12800:
            device_per_token_quant_bf16_to_fp8<256, 12800>
            <<<blocks, 256, 0, at::cuda::getCurrentCUDAStream()>>>(
                PTR<bf16_t>(contiguous_input),
                PTR<fp8_e4m3_t>(output),
                PTR<fp32_t>(contiguous_scales)
            );
            break;
        default: {
            static constexpr int TPB = 128;
            const int64_t shared_mem_size = N * sizeof(bf16_t);
            if (N % 8 == 0) {
                device_per_token_quant_bf16_to_fp8_vpt<TPB>
                <<<blocks, TPB, shared_mem_size, at::cuda::getCurrentCUDAStream()>>>(
                    PTR<bf16_t>(contiguous_input),
                    PTR<fp8_e4m3_t>(output),
                    PTR<fp32_t>(contiguous_scales),
                    N
                );
            } else {
                device_per_token_quant_bf16_to_fp8_general<TPB>
                <<<blocks, TPB, shared_mem_size, at::cuda::getCurrentCUDAStream()>>>(
                    PTR<bf16_t>(contiguous_input),
                    PTR<fp8_e4m3_t>(output),
                    PTR<fp32_t>(contiguous_scales),
                    N
                );
            }
        }
    }

    return;
}

} // namespace ops
} // namespace lightllm
